#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2014, NVIDIA Corporation
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * * Redistributions of source code must retain the above copyright
 *   notice, this list of conditions and the following disclaimer.
 * * Redistributions in binary form must reproduce the above copyright
 *   notice, this list of conditions and the following disclaimer in the
 *   documentation and/or other materials provided with the distribution.
 * * Neither the name of the NVIDIA Corporation nor the
 *   names of its contributors may be used to endorse or promote products
 *   derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 **/

#include "gpu_vector.h"
#include <thrust/reduce.h>
#include <thrust/inner_product.h>
#include <thrust/transform_reduce.h>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <sstream>

#include <hipblas.h>

hipblasHandle_t blas_handle = NULL;

#ifdef __cplusplus
extern "C" {
#endif

#define FatalError(s) {                                                 \
  std::stringstream _where;                                             \
  _where << __FILE__ << ':' << __LINE__;                                \
  printf("[E]: %s at %s\n",s.c_str(),_where.str().c_str());             \
  hipDeviceSynchronize();                                              \
  exit(0);                                                              \
}

// device-sync and check error only if in debug mode
#if !defined(NDEBUG)
#define cudaCheckError() {                                              \
  hipDeviceSynchronize();                                              \
  hipError_t e=hipGetLastError();                                     \
  if(e!=hipSuccess) {                                                  \
    std::stringstream _error;                                           \
    _error << "Cuda failure: '" << hipGetErrorString(e) << "'";        \
    FatalError(_error.str());                                           \
  }                                                                     \
}
#else
#define cudaCheckError() {}
#endif

#define CUCHK(call) {                                    \
  hipError_t err = call;                                                    \
  if( hipSuccess != err) {                                                \
  fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
          __FILE__, __LINE__, hipGetErrorString( err) );              \
  fflush(stderr); \
  exit(EXIT_FAILURE);                                                  \
  } }

__global__
void kernel_SetConstantValue(double value, double *data, int size)
{
  for (int idx=threadIdx.x+blockIdx.x*blockDim.x; idx<size; idx+=gridDim.x*blockDim.x) {
    data[idx] = value;
  }
}

void device_checkErrors()
{
  cudaCheckError();
  CUCHK(hipGetLastError());
}

void device_createCublas()
{
  hipblasStatus_t status;
  if (blas_handle == NULL) {
    status = hipblasCreate(&blas_handle);
  }
  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf("[E]: hipblasCreate in %s failed (%d)\n",__FUNCTION__,(int)status);
  }
  cudaCheckError();
}

double device_VectorSumElts(double *data, int size)
{
  double sum = thrust::reduce(data, data+size);
  cudaCheckError();
  return sum;
}

void device_SeqVectorScale(double alpha, double *data, int size)
{
  hipblasStatus_t status;
  if (blas_handle == NULL) {
    status = hipblasCreate(&blas_handle);
  }

  status = hipblasDscal(blas_handle, size, &alpha, data, 1);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf("[E]: hipblasDscal in device_SeqVectorScale failed (%d)\n",(int)status);
  }
  cudaCheckError();
}

void device_SeqVectorAxpy(double alpha, double *x, double *y, int size)
{
  hipblasStatus_t status;
  if (blas_handle == NULL) {
    status = hipblasCreate(&blas_handle);
  }

  status = hipblasDaxpy(blas_handle, size, &alpha, x, 1, y, 1);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf("[E]: hipblasDaxpy in device_SeqVectorAxpy failed (%d)\n",(int)status);
  }
  cudaCheckError();
}

double device_SeqVectorInnerProd(double *x, double *y, int size)
{
  hipblasStatus_t status;
  if (blas_handle == NULL) {
    status = hipblasCreate(&blas_handle);
  }

  double r = 0.;

  // r = thrust::inner_product(thrust::device, x, x+size, y, 0.0);
  status = hipblasDdot(blas_handle, size, x, 1, y, 1, &r);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf("[E]: hipblasDdot in device_SeqVectorInnerProd failed (%d)\n",(int)status);
  }
  cudaCheckError();

  return r;
} 

void device_SeqVectorCopy(double *x, double *y, int size)
{
  hipblasStatus_t status;
  if (blas_handle == NULL) {
    status = hipblasCreate(&blas_handle);
  }

  // copy x into y
  status = hipblasDcopy(blas_handle, size, x, 1, y, 1);

  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf("[E]: hipblasDcopy in device_SeqVectorCopy failed (%d)\n",(int)status);
  }
  cudaCheckError();
}

void device_SeqVectorSetConstantValues(double value, double *data, int size)
{
  kernel_SetConstantValue<<<4096, 512>>>(value, data, size);
  CUCHK(hipGetLastError());
  cudaCheckError();
}

#ifdef __cplusplus
}; // end extern "C"
#endif
